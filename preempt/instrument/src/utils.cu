#include "hip/hip_runtime.h"
#include <fstream>

#include "nvbit.h"
#include "instrument.h"

void save_to_file(std::string filename, char *buf, size_t size)
{
    std::ofstream out(filename, std::ios::binary);
    out.write(buf, size);
    out.flush();
    out.close();
}

__global__ void memcpyDtoD(void *dst_device, void *src_device, size_t size)
{
    memcpy(dst_device, src_device, size);
}

void memcpyDtoH_force(void *dst_host, const hipDeviceptr_t src_device, const size_t size, hipStream_t stream)
{
    hipDeviceptr_t readable_devptr;
    ASSERT_GPU_ERROR(hipMallocAsync(&readable_devptr, size, stream));
    ASSERT_GPU_ERROR(hipStreamSynchronize(stream));
    memcpyDtoD<<<1, 1, 0, stream>>>((void *)readable_devptr, (void *)src_device, size);
    ASSERT_GPU_ERROR(hipMemcpyDtoHAsync(dst_host, readable_devptr, size, stream));
    ASSERT_GPU_ERROR(hipFreeAsync(readable_devptr, stream));
    ASSERT_GPU_ERROR(hipStreamSynchronize(stream));
}

void memcpyHtoD_force(hipDeviceptr_t dst_device, const void *src_host, const size_t size, hipStream_t stream)
{
    hipDeviceptr_t readable_devptr;
    ASSERT_GPU_ERROR(hipMallocAsync(&readable_devptr, size, stream));
    ASSERT_GPU_ERROR(hipStreamSynchronize(stream));
    ASSERT_GPU_ERROR(hipMemcpyHtoDAsync(readable_devptr, src_host, size, stream));
    memcpyDtoD<<<1, 1, 0, stream>>>((void *)dst_device, (void *)readable_devptr, size);
    ASSERT_GPU_ERROR(hipFreeAsync(readable_devptr, stream));
    ASSERT_GPU_ERROR(hipStreamSynchronize(stream));
}

void memcpyDtoHAsync_force(void *dst_host, const hipDeviceptr_t src_device, const hipDeviceptr_t device_buffer, const size_t size, hipStream_t stream)
{
    memcpyDtoD<<<1, 1, 0, stream>>>((void *)device_buffer, (void *)src_device, size);
    ASSERT_GPU_ERROR(hipMemcpyDtoHAsync(dst_host, device_buffer, size, stream));
}

void memcpyHtoDAsync_force(hipDeviceptr_t dst_device, const void *src_host, const hipDeviceptr_t device_buffer, const size_t size, hipStream_t stream)
{
    ASSERT_GPU_ERROR(hipMemcpyHtoDAsync(device_buffer, src_host, size, stream));
    memcpyDtoD<<<1, 1, 0, stream>>>((void *)dst_device, (void *)device_buffer, size);
}
